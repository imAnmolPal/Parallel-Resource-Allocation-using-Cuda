
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define max_N 100000
#define max_P 30
#define BLOCKSIZE 1024

using namespace std;

//*******************************************

// Write down the kernels here

/* Before you go into the code, one good job that has been done in this assignment is 
that I have done sorting in O(R) time, instead of O(R*ln(R)).

For that I have exploited the fact that the elements that we have to sort are in range of 
of 1 to N-1 and used prefix sum to have them inorder(computer centre wise)

I am pretty sure, this is a great optimization on the part of preprocessing without using 
external library or traditional sorting algorithms. */


/*Here in this kernel we have reduced the processing of the number of request per computer centre.
Each computer center will process only those many request which are assigned to them.

We took help of sorting to map request to their corresponding computer centre.

One optimization could also have been done here, we can call another kernel from this kernel instead of the 
loop that are running inside the kernel, that will optimize to some extent.

As there are no dependency within the loops, so parallelism could have be exploitted. */
__global__ void optimizedK(int *reqCountForEachComCen, int *exclusivePrefixSum, int *tot_reqs, int *succ_reqs,
 int *capacity, int *req_cen,int *req_fac, int *req_start, int *req_slots, int *fac_prefixSum){
  
  /*This comNum gives us the computer center number*/
  int comNum = blockIdx.x;

  /* For bug testing */
  // printf("ComCen - %d , ID Range - from %d to %d places \n", comNum, exclusivePrefixSum[comNum], reqCountForEachComCen[comNum]);
  // printf("")

  /* Using hours array to have hold of the fact that which slots are available and 
  which slots are occupied. */
  int hours[30*24+1];
  for(int i=0;i<30*24+1;i++){
    hours[i]=0;
  }

  /*This start and end points helps us to find the mapping of all the request to Computer Centre comNum. */
  int start = exclusivePrefixSum[comNum];
  int end = exclusivePrefixSum[comNum]+reqCountForEachComCen[comNum]-1;


  /* Processing the request. Loops inside this loop could be further avoided by the help
  of dynamic parallelism. */
  for(int i=start; i<=end; i++){
    tot_reqs[comNum]++;


    bool flag = false;
    // int facIndex = exclusivePrefixSum[comNum]+req_fac[i];
    
    /*To know about capacity of the required facility using actual index of that facility.
    Which we find out using prefix sum.*/
    int facIndex = fac_prefixSum[req_cen[i]]+req_fac[i];
    int capacityOfReqFacility = capacity[facIndex];
    
    /* For debugging purpose. */
    // if(i==0){
    //   printf("\n Capacity of Req Facility: %d \n", capacityOfReqFacility);
    // }


    /*traverse hours array to check if slot available*/
    int start_Hour = req_start[i];
    int runTill = req_start[i]+req_slots[i];

    /* Here are checking if slot is available or not with the help of flag variable.*/
    for(int j=req_fac[i]*24+start_Hour; j<req_fac[i]*24+runTill;j++){
      if(hours[j]<capacityOfReqFacility){
        continue;
      }else{
        // cc[req_cen[i]*2+1]+=1;
        flag = true;
        break;
      }
    }

    /* If slots are not availablle then switch to next request.*/
    if(flag==true){
      continue;
    }

    /* If slots are available then process the request and increase the success
    count for computer centre "comNum" */
    for(int j=req_fac[i]*24+start_Hour; j<req_fac[i]*24+runTill;j++){
            hours[j]++;
    }
    succ_reqs[comNum]++;
  }
}


//***********************************************


int main(int argc,char **argv)
{
	// variable declarations...
    int N,*centre,*facility,*capacity,*fac_ids, *succ_reqs, *tot_reqs;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &N ); // N is number of centres
	
    // Allocate memory on cpu
    centre=(int*)malloc(N * sizeof (int));  // Computer  centre numbers
    facility=(int*)malloc(N * sizeof (int));  // Number of facilities in each computer centre
    fac_ids=(int*)malloc(max_P * N  * sizeof (int));  // Facility room numbers of each computer centre
    capacity=(int*)malloc(max_P * N * sizeof (int));  // stores capacities of each facility for every computer centre 


    int success=0;  // total successful requests
    int fail = 0;   // total failed requests
    tot_reqs = (int *)malloc(N*sizeof(int));   // total requests for each centre
    succ_reqs = (int *)malloc(N*sizeof(int)); // total successful requests for each centre

    // Input the computer centres data
    int k1=0 , k2 = 0;
    for(int i=0;i<N;i++)
    {
      fscanf( inputfilepointer, "%d", &centre[i] );
      fscanf( inputfilepointer, "%d", &facility[i] );
      
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &fac_ids[k1] );
        k1++;
      }
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &capacity[k2]);
        k2++;     
      }
    }

    // variable declarations
    int *req_id, *req_cen, *req_fac, *req_start, *req_slots;   // Number of slots requested for every request
    
    // Allocate memory on CPU 
	int R;
	fscanf( inputfilepointer, "%d", &R); // Total requests
    req_id = (int *) malloc ( (R) * sizeof (int) );  // Request ids
    req_cen = (int *) malloc ( (R) * sizeof (int) );  // Requested computer centre
    req_fac = (int *) malloc ( (R) * sizeof (int) );  // Requested facility
    req_start = (int *) malloc ( (R) * sizeof (int) );  // Start slot of every request
    req_slots = (int *) malloc ( (R) * sizeof (int) );   // Number of slots requested for every request
    
    // Input the user request data
    for(int j = 0; j < R; j++)
    {
       fscanf( inputfilepointer, "%d", &req_id[j]);
       fscanf( inputfilepointer, "%d", &req_cen[j]);
       fscanf( inputfilepointer, "%d", &req_fac[j]);
       fscanf( inputfilepointer, "%d", &req_start[j]);
       fscanf( inputfilepointer, "%d", &req_slots[j]);
       tot_reqs[req_cen[j]]+=1;  
    }
		
    //********************************* 


    /* To have the request in sorted form.*/
    int *sReqID, *sReqCen, *sReqFac, *sReqStart, *sReqSlots;
    sReqID = (int *) malloc ((R) * sizeof (int));
    sReqCen = (int *) malloc ((R) * sizeof (int));
    sReqFac = (int *) malloc ((R) * sizeof (int));
    sReqStart = (int *) malloc ((R) * sizeof (int));
    sReqSlots = (int *) malloc ((R) * sizeof (int));



    /* To know individual count of request for each computer. */
    int *reqCountForEachComCen;
    reqCountForEachComCen = (int*)malloc(N * sizeof (int));
    memset(reqCountForEachComCen, 0, N*sizeof(int));

    for(int i=0; i<R; i++){
      reqCountForEachComCen[req_cen[i]]++;
    }

    /* We are using iterator to sort the request. exclusivePRefixSum is 
    sum of capacity which is also helping us in sorting.*/
    int *iterator, *exclusivePrefixSum;

    /*Preprocessing for the sorting.*/
    iterator = (int*)malloc(N * sizeof (int));
    exclusivePrefixSum = (int*)malloc(N * sizeof (int));

    iterator[0]=0;
    exclusivePrefixSum[0]=0;

    for(int i=1;i<N;i++){
      iterator[i] = reqCountForEachComCen[i-1]+iterator[i-1];
      exclusivePrefixSum[i]= iterator[i];
    }

    /* Sorting the request in O(R) time. */
    for(int i=0;i<R;i++){
      int sortedIndex = iterator[req_cen[i]];
      sReqID[sortedIndex]=req_id[i];
      sReqCen[sortedIndex]=req_cen[i];
      sReqFac[sortedIndex]=req_fac[i];
      sReqStart[sortedIndex]=req_start[i];
      sReqSlots[sortedIndex]=req_slots[i];
      iterator[req_cen[i]]++;
    }


    /*For debugging...*/
    // for(int i=0;i<R;i++){
    //   printf("%d %d %d %d %d \n", sReqID[i], sReqCen[i], sReqFac[i], sReqStart[i], sReqSlots[i]);
    // }

    /*Allocating memory to device.*/
    int *d_capacity, *d_sReqCen,*d_sReqFac, *d_sReqStart, *d_sReqSlots;
    hipMalloc(&d_capacity,max_P * N*sizeof(int));
    hipMalloc(&d_sReqCen,R*sizeof(int));
    hipMalloc(&d_sReqFac,R*sizeof(int));
    hipMalloc(&d_sReqStart,R*sizeof(int));
    hipMalloc(&d_sReqSlots,R*sizeof(int));

    hipMemcpy(d_capacity, capacity, max_P * N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sReqCen, sReqCen, R*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sReqFac, sReqFac, R*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sReqStart, sReqStart, R*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sReqSlots, sReqSlots, R*sizeof(int), hipMemcpyHostToDevice);




    int *d_reqCountForEachComCen;
    hipMalloc(&d_reqCountForEachComCen,N*sizeof(int));
    hipMemcpy(d_reqCountForEachComCen, reqCountForEachComCen, N*sizeof(int), hipMemcpyHostToDevice);
    
    int *d_exclusivePrefixSum;
    hipMalloc(&d_exclusivePrefixSum,N*sizeof(int));
    hipMemcpy(d_exclusivePrefixSum, exclusivePrefixSum, N*sizeof(int), hipMemcpyHostToDevice);

    // total and successful requests 
    int *d_tot_reqs;
    hipMalloc(&d_tot_reqs,N*sizeof(int));
    hipMemset(d_tot_reqs, 0, N*sizeof(int));
  
    int *d_succ_reqs;
    hipMalloc(&d_succ_reqs,N*sizeof(int));
    // cudaMemcpy(d_succ_reqs, succ_reqs, N*sizeof(int), cudaMemcpyHostToDevice);
    hipMemset(d_succ_reqs, 0, N*sizeof(int));


    /* Taking prefix array sum of the facility which will help us in 
    finding effective index of facility in kernel.*/
    
    int *prefixArray; //
    prefixArray = (int *) malloc(N*sizeof(int)); // prefix array to know capacity of indivual facility
    /* find prefix sum */ 
    int temp = 0;
    for(int i=0;i<N;i++){
      prefixArray[i]=temp;
      temp+=facility[i];
    }


    /* Debugging ...*/
    // for(int i=0;i<N;i++){
    //   printf("%d ", prefixArray[i]);
    // }
    // // d_prefixSum

    /* Allocating the memory in device to prefixSum array.*/
    int *d_prefixSum;
    hipMalloc(&d_prefixSum, N*sizeof(int));
    hipMemcpy(d_prefixSum, prefixArray, N*sizeof(int), hipMemcpyHostToDevice);


    /* Launching the kernel...*/
    int BLOCK = N;
    int THREAD = 1;
    optimizedK<<<BLOCK, THREAD>>>(d_reqCountForEachComCen, d_exclusivePrefixSum, d_tot_reqs, d_succ_reqs,
     d_capacity, d_sReqCen,d_sReqFac, d_sReqStart, d_sReqSlots, d_prefixSum);
    
    hipDeviceSynchronize();

    hipMemcpy(succ_reqs, d_succ_reqs, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(tot_reqs, d_tot_reqs, N*sizeof(int), hipMemcpyDeviceToHost);
    for(int i=0;i<N;i++){
      success+=succ_reqs[i];
    }
    // printf("Kernel End\n Total Req Processed: %d", tot_reqs[0]);
    fail = R - success; 

    /*Debugging...*/
    // printf("%d %d", success, fail);
    //********************************

    //*********************************
    // Call the kernels here

    //********************************




    // Output
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    fprintf( outputfilepointer, "%d %d\n", success, fail);
    for(int j = 0; j < N; j++)
    {
        fprintf( outputfilepointer, "%d %d\n", succ_reqs[j], tot_reqs[j]-succ_reqs[j]);
    }
    fclose( inputfilepointer );
    fclose( outputfilepointer );
    hipDeviceSynchronize();
	return 0;
}